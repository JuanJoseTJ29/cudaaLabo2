#include "hip/hip_runtime.h"
#include <thread>
#include <time.h>
#include <iostream>
#include <math.h>
#include "imageLoader.cpp"

#define GRIDVAL 20.0

__global__ void sobel_gpu(const byte *orig, byte *cpu, const unsigned int width, const unsigned int height)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    float dx, dy;
    if (x > 0 && y > 0 && x < width - 1 && y < height - 1)
    {
        dx = (-1 * orig[(y - 1) * width + (x - 1)]) + (-2 * orig[y * width + (x - 1)]) + (-1 * orig[(y + 1) * width + (x - 1)]) +
             (orig[(y - 1) * width + (x + 1)]) + (2 * orig[y * width + (x + 1)]) + (orig[(y + 1) * width + (x + 1)]);
        dy = (orig[(y - 1) * width + (x - 1)]) + (2 * orig[(y - 1) * width + x]) + (orig[(y - 1) * width + (x + 1)]) +
             (-1 * orig[(y + 1) * width + (x - 1)]) + (-2 * orig[(y + 1) * width + x]) + (-1 * orig[(y + 1) * width + (x + 1)]);
        cpu[y * width + x] = sqrt((dx * dx) + (dy * dy));
    }
}

int main(int argc, char *argv[])
{

    imgData origImg = loadImage(argv[1]);
    imgData gpuImg(new byte[origImg.width * origImg.height], origImg.width, origImg.height);

    byte *gpu_orig, *gpu_sobel;
    hipMalloc((void **)&gpu_orig, (origImg.width * origImg.height));
    hipMalloc((void **)&gpu_sobel, (origImg.width * origImg.height));

    hipMemcpy(gpu_orig, origImg.pixels, (origImg.width * origImg.height), hipMemcpyHostToDevice);
    hipMemset(gpu_sobel, 0, (origImg.width * origImg.height));

    dim3 threadsPerBlock(GRIDVAL, GRIDVAL, 1);
    dim3 numBlocks(ceil(origImg.width / GRIDVAL), ceil(origImg.height / GRIDVAL), 1);

    sobel_gpu<<<numBlocks, threadsPerBlock>>>(gpu_orig, gpu_sobel, origImg.width, origImg.height);
    hipError_t cudaerror = hipDeviceSynchronize();

    if (cudaerror != hipSuccess)
        fprintf(stderr, "Cuda failed to synchronize: %s\n", hipGetErrorName(cudaerror));

    hipMemcpy(gpuImg.pixels, gpu_sobel, (origImg.width * origImg.height), hipMemcpyDeviceToHost);

    writeImage(argv[1], "gpu", gpuImg);

    hipFree(gpu_orig);
    hipFree(gpu_sobel);
    return 0;
}
